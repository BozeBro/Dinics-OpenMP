#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "dinics-seq.hpp"

__global__ void bfsKernel(char* frontier, char* newFrontier, int level, 
    int* edgesStart, int* edgesCount, int* edges, int* layeredEdgesCount, 
    int* layeredEdges, int* edgeCapacities, unsigned int* vertDists, 
    int count, char* foundSink, char* progressed) {
  int vert = blockIdx.x * blockDim.x + threadIdx.x;

  if (!frontier[vert] || vert >= count)
    return;

  int es = edgesStart[vert];
  int ec = edgesCount[vert];
  int dist = level;
  int lc = 0;

  for (int i = 0; i < ec; i++) {
    int dest = edges[es + i];
    
    if (edgeCapacities[es + i] > 0 && vertDists[dest] >= dist + 1) {
      if (dest == 1)
        *foundSink = true;

      vertDists[dest] = dist + 1;
      newFrontier[dest] = true;
      layeredEdges[es + lc] = dest;
      *progressed = true;
      lc++;
    }
  }

  layeredEdgesCount[vert] = lc;
}

bool Graph::bfsCuda() {
  char* cudaFrontier;
  hipMalloc(&cudaFrontier, sizeof(char) * vertices.size());
  hipMemset(cudaFrontier, 0, sizeof(char) * vertices.size());
  hipMemset(cudaFrontier, 1, sizeof(char));

  char* cudaNewFrontier;
  hipMalloc(&cudaNewFrontier, sizeof(char) * vertices.size());
  hipMemset(cudaNewFrontier, 0, sizeof(char) * vertices.size());

  int* cudaEdgesStart;
  hipMalloc(&cudaEdgesStart, sizeof(int) * vertices.size());

  int* cudaEdgesCount;
  hipMalloc(&cudaEdgesCount, sizeof(int) * vertices.size());

  int* cudaEdges;
  hipMalloc(&cudaEdges, sizeof(int) * edgeCount);

  int* cudaLayeredEdgesCount;
  hipMalloc(&cudaLayeredEdgesCount, sizeof(int) * vertices.size());

  int* cudaLayeredEdges;
  hipMalloc(&cudaLayeredEdges, sizeof(int) * edgeCount);

  int* cudaEdgeCapacities;
  hipMalloc(&cudaEdgeCapacities, sizeof(int) * edgeCount);

  unsigned int* cudaVertDists;
  hipMalloc(&cudaVertDists, sizeof(int) * vertices.size());
  hipMemset(cudaVertDists, 255, sizeof(int) * vertices.size());
  hipMemset(cudaVertDists, 0, sizeof(int));

  char* cudaFoundSink;
  hipMalloc(&cudaFoundSink, sizeof(char));
  hipMemset(cudaFoundSink, 0, sizeof(char));

  char* cudaProgressed;
  hipMalloc(&cudaProgressed, sizeof(char));

  std::vector<int> edges(edgeCount);
  std::vector<int> edgeCapacities(edgeCount);
  std::vector<int> edgesStart(vertices.size());
  std::vector<int> edgesCount(vertices.size());

  int ec = 0;
  for (int i = 0; i < vertices.size(); i++) {
    edgesStart[i] = ec;
    int numNeighbors = neighbors[i].size();
    edgesCount[i] = numNeighbors;

    int e = 0;
    for (auto &[dst, edge] : this->neighbors[i]) {
      edgeCapacities[ec + e] = edge.cap;
      edges[ec + e] = dst;
      e++;
    }
    ec += e;
  }

  int level = 0;
  hipMemcpy(cudaEdges, edges.data(), edgeCount * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaEdgeCapacities, edgeCapacities.data(), edgeCount * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaEdgesStart, edgesStart.data(), vertices.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaEdgesCount, edgesCount.data(), vertices.size() * sizeof(int), hipMemcpyHostToDevice);

  const int threadsPerBlock = 512;
  const int blocks = (vertices.size() + threadsPerBlock - 1) / threadsPerBlock;

  char foundSink = false;
  char progressed = false;

  do {
    hipMemset(cudaProgressed, 0, sizeof(char));
    bfsKernel<<<blocks, threadsPerBlock>>>(cudaFrontier, cudaNewFrontier, level, 
      cudaEdgesStart, cudaEdgesCount, cudaEdges, cudaLayeredEdgesCount, 
      cudaLayeredEdges, cudaEdgeCapacities, cudaVertDists, 
      vertices.size(), cudaFoundSink, cudaProgressed);

    hipMemcpy(cudaFrontier, cudaNewFrontier, sizeof(char) * vertices.size(), hipMemcpyDeviceToDevice);
    hipMemset(cudaNewFrontier, 0, sizeof(char) * vertices.size());

    hipMemcpy(&foundSink, cudaFoundSink, sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(&progressed, cudaProgressed, sizeof(char), hipMemcpyDeviceToHost);

    level++;
  } while (!foundSink && progressed);

  std::vector<int> layeredEdgesCount(vertices.size());
  std::vector<int> layeredEdges(edgeCount);
  hipMemcpy(layeredEdges.data(), cudaLayeredEdges, edgeCount * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(layeredEdgesCount.data(), cudaLayeredEdgesCount, vertices.size() * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < vertices.size(); i++) {
    vertices[i].layered_dst_array[0].resize(layeredEdgesCount[i]);
    for (int j = 0; j < layeredEdgesCount[i]; j++) {
      vertices[i].layered_dst_array[0][j] = layeredEdges[edgesStart[i] + j];
    }
  }

  hipFree(cudaFrontier);
  hipFree(cudaNewFrontier);
  hipFree(cudaEdgesStart);
  hipFree(cudaEdgesCount);
  hipFree(cudaEdges);
  hipFree(cudaLayeredEdgesCount);
  hipFree(cudaLayeredEdges);
  hipFree(cudaEdgeCapacities);
  hipFree(cudaVertDists);
  hipFree(cudaFoundSink);

  return foundSink;
}

