#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "dinics-seq.hpp"
#include <chrono>

__global__ void bfsKernel(char* frontier, char* newFrontier, int level, 
    int* edgesStart, int* edgesCount, int* edges, int* layeredEdgesCount, 
    int* layeredEdges, int* edgeCapacities, unsigned int* vertDists, 
    int count, char* foundSink, char* progressed) {
  int vert = blockIdx.x * blockDim.x + threadIdx.x;

  if (!frontier[vert] || vert >= count)
    return;

  int es = edgesStart[vert];
  int ec = edgesCount[vert];
  int dist = level;
  int lc = 0;

  for (int i = 0; i < ec; i++) {
    int dest = edges[es + i];
    
    if (edgeCapacities[es + i] > 0 && vertDists[dest] >= dist + 1) {
      if (dest == 1)
        *foundSink = true;

      vertDists[dest] = dist + 1;
      newFrontier[dest] = true;
      layeredEdges[es + lc] = dest;
      *progressed = true;
      lc++;
    }
  }

  layeredEdgesCount[vert] = lc;
}

void Graph::initCuda() {
  hipMalloc(&cudaFrontier, sizeof(char) * vertices.size());
  hipMalloc(&cudaNewFrontier, sizeof(char) * vertices.size());
  hipMalloc(&cudaEdgesStart, sizeof(int) * vertices.size());
  hipMalloc(&cudaEdgesCount, sizeof(int) * vertices.size());
  hipMalloc(&cudaEdges, sizeof(int) * edgeCount);
  hipMalloc(&cudaLayeredEdgesCount, sizeof(int) * vertices.size());
  hipMalloc(&cudaLayeredEdges, sizeof(int) * edgeCount);
  hipMalloc(&cudaEdgeCapacities, sizeof(int) * edgeCount);
  hipMalloc(&cudaVertDists, sizeof(int) * vertices.size());
  hipMalloc(&cudaFoundSink, sizeof(char));
  hipMalloc(&cudaProgressed, sizeof(char));

  hipMemcpy(cudaEdges, edges.data(), edgeCount * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaEdgesStart, edgesStart.data(), vertices.size() * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaEdgesCount, edgesCount.data(), vertices.size() * sizeof(int), hipMemcpyHostToDevice);
}

void Graph::destroyCuda() {
  hipFree(cudaFrontier);
  hipFree(cudaNewFrontier);
  hipFree(cudaEdgesStart);
  hipFree(cudaEdgesCount);
  hipFree(cudaEdges);
  hipFree(cudaLayeredEdgesCount);
  hipFree(cudaLayeredEdges);
  hipFree(cudaEdgeCapacities);
  hipFree(cudaVertDists);
  hipFree(cudaFoundSink);
}

bool Graph::bfsCuda() {
  hipMemset(cudaFrontier, 0, sizeof(char) * vertices.size());
  hipMemset(cudaFrontier, 1, sizeof(char));
  hipMemset(cudaNewFrontier, 0, sizeof(char) * vertices.size());
  hipMemset(cudaVertDists, 255, sizeof(int) * vertices.size());
  hipMemset(cudaVertDists, 0, sizeof(int));
  hipMemset(cudaFoundSink, 0, sizeof(char));

  // std::vector<int> edges(edgeCount);
  // std::vector<int> edgeCapacities(edgeCount);
  // std::vector<int> edgesStart(vertices.size());
  // std::vector<int> edgesCount(vertices.size());

  auto start = std::chrono::steady_clock::now();
  // int ec = 0;
  // for (int i = 0; i < vertices.size(); i++) {
  //   edgesStart[i] = ec;
  //   int numNeighbors = neighbors[i].size();
  //   edgesCount[i] = numNeighbors;

  //   int e = 0;
  //   for (auto &[dst, edge] : this->neighbors[i]) {
  //     edgeCapacities[ec + e] = edge.cap;
  //     edges[ec + e] = dst;
  //     e++;
  //   }
  //   ec += e;
  // }

  int level = 0;
  hipMemcpy(cudaEdgeCapacities, edgeCapacities.data(), edgeCount * sizeof(int), hipMemcpyHostToDevice);
  auto end = std::chrono::steady_clock::now();

  const int threadsPerBlock = 512;
  const int blocks = (vertices.size() + threadsPerBlock - 1) / threadsPerBlock;

  char foundSink = false;
  char progressed = false;
  bfs_aux_time +=
      std::chrono::duration_cast<std::chrono::duration<double>>(end - start)
          .count();

  do {
    hipMemset(cudaProgressed, 0, sizeof(char));
    bfsKernel<<<blocks, threadsPerBlock>>>(cudaFrontier, cudaNewFrontier, level, 
      cudaEdgesStart, cudaEdgesCount, cudaEdges, cudaLayeredEdgesCount, 
      cudaLayeredEdges, cudaEdgeCapacities, cudaVertDists, 
      vertices.size(), cudaFoundSink, cudaProgressed);

    hipMemcpy(cudaFrontier, cudaNewFrontier, sizeof(char) * vertices.size(), hipMemcpyDeviceToDevice);
    hipMemset(cudaNewFrontier, 0, sizeof(char) * vertices.size());

    hipMemcpy(&foundSink, cudaFoundSink, sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(&progressed, cudaProgressed, sizeof(char), hipMemcpyDeviceToHost);

    level++;
  } while (!foundSink && progressed);

  hipMemcpy(layeredEdges.data(), cudaLayeredEdges, edgeCount * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(layeredEdgesCount.data(), cudaLayeredEdgesCount, vertices.size() * sizeof(int), hipMemcpyDeviceToHost);
  // for (int i = 0; i < vertices.size(); i++) {
  //   vertices[i].layered_dst.resize(layeredEdgesCount[i]);
  //   for (int j = 0; j < layeredEdgesCount[i]; j++) {
  //     vertices[i].layered_dst[j] = layeredEdges[edgesStart[i] + j];
  //   }
  // }

  return foundSink;
}

